#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>




// Anzahl Threads pro Blockdimension
const int blocksize = 32;




/////////////////////////////////////////////////////////////////
//                       Hilfsfunktionen                       //
/////////////////////////////////////////////////////////////////


/* Funktion zur Ausgabe eines Vektors. */
void print_vec (double *v, int dim, char *name)
{
        printf("\n%s:\n", name);
        for (int i = 0; i < dim; i++)
        {
                printf("% lf\n" , v[i]);
        }
        printf("\n");
}


/* Funktion zum Ueberfuehren eines Vektors auf den inneren Gitterpunkten
 auf das Gitter mit Ghost Layer. */
void ghostify (double *v, double *v_ghost, int N)
{
    int k; // Gitterpunkt des Gitteers mit Ghost Layer
    int ind = 0; // Aktueller Eintrag des Ausgangsvektors

    // Durchgehen des Gitters mit Ghost Layer
    for (int j=1; j <= N+2; j++) // Gitterzeile
    {
        for (int i=1; i <= N+2; i++) // Gitterspalte
        {
            k = (j-1)*(N+2)+(i-1); // Gitterpunkt

            if ( (k < N+2) || (k%(N+2) == 0) || (k%(N+2) == N+1) || (k >= (N+1)*(N+2)) ) // Gitterpunkt faellt in Ghost Layer
            {
                v_ghost[k] = 0;
            }
            else
            {
                v_ghost[k] = v[ind]; // Gitterpunkt ist innerer Gitterpunkt
                ind += 1;
            }
        }
    }
}


/* Funktion zum Ueberfuehren eines Vektors auf dem Gitter mit Ghost Layer
 auf die inneren Gitterpunkte. */
void deghostify (double *v_ghost, double *v, int N)
{
    int k; // Gitterpunkt des Gitters mit Ghost Layer
    int ind = 0; // Aktueller Eintrag des Zielvektors

    // Durchgehen des Gitters mit Ghost Layer nur auf den inneren Punkten
    for (int j=2; j <= N+1; j++) // Gitterzeile
    {
        for (int i=2; i <= N+1; i++) // Gitterspalte
        {
            k = (j-1)*(N+2) + (i-1); // Gitterpunkt

            v[ind] = v_ghost[k];
            ind += 1;
        }
    }
}


/* Funktion zur Bestimmung des Skalarprodukts zweier Vektoren fester Laenge. */
double dot (double *v1, double *v2, int dim)
{
    double res = 0; // Ergebnis

        for(int i = 0; i < dim; i++)
        {
                res += v1[i]*v2[i];
        }

        return res;
}


/* Funktion zur Berechnung der euklidischen Norm eines Vektors. */
double norm_cpu (double *v, int dim)
{
        double res = sqrt(dot(v,v,dim));
        return res;
}


/* Funktion zur Berechnung der Wurzel von einem Vektor mit einem Eintrag. */
void sq (double *v)
{
        v[0]=sqrt(v[0]);
}

/* Funktion zur Bestimmung des absoluten Fehlers der Naeherung eines Loesungsvektors als Norm der Differenz
 zwischen der Naeherung und der Loesung. */
double get_abs_err (double *u, double *u_exakt, int dim)
{
        // Differenzvektor
        double *diff = (double *)malloc(dim*sizeof(double));

        for (int i = 0; i < dim; i++)
        {
                diff[i] = u[i] - u_exakt[i];
        }

        // Absoluter Fehler als Norm des Differenzvektors
        double abs_err = norm_cpu(diff,dim);

        free(diff);

        return abs_err;
}

/////////////////////////////////////////////////////////////////
//            Funktionen zum Befuellen von Vektoren            //
/////////////////////////////////////////////////////////////////


/* Funktion zum Befuellen eines Vektors mit der exakten Loesung des Poisson-Problems an
 den inneren Gitterpunkten nach zeilenweiser Nummerierung. */
void get_solution (double *u_exakt, int N)
{
        // Schrittweite
        double h = 1/(double)(N+1);

        // Durchgehen des Gitters
        for (int j=1; j<=N; j++) // Gitterzeile
        {
                for (int i=1; i<=N; i++) // Gitterspalte
                {
                        u_exakt[(i-1)+(j-1)*N]=sin(M_PI*i*h)*sin(M_PI*j*h);
                }
        }
}


/* Funktion zum Befuellen eines Vektors mit der Auswertung der gegebenen rechten Seite des
 Poissons-Problems an den inneren Gitterpunkten nach zeilenweiser Nummerierung. */
void get_RHS (double *f, int N)
{
        // Schrittweite
        double h = 1/(double)(N+1);

        // Durchgehen des Gitters
        for (int j=1; j<=N; j++) // Gitterzeile
        {
                for (int i=1; i<=N; i++) // Gitterspalte
                {
                        f[(i-1)+(j-1)*N]=h*h*2*M_PI*M_PI*sin(M_PI*i*h)*sin(M_PI*j*h); // h^2 auf rechter Seite
                }
        }
}


/* Funktion zum Befuellen eines Vektors mit zufaelligen Eintraegen in einem spezifizierten Wertebereich. */
void get_random_vec (double *v, int dim, double min, double max)
{
        // Setzen eines Random-Seeds
        srand(time(0));

        // Fuellen des Vektors mit zufaelligen Eintraegen
        for (int i = 0; i < dim; i++)
        {
                v[i] = min + ((double)rand()/(double)RAND_MAX) * (max - min);
        }
}


/* Funktion zum Befuellen eines Vektors als Startvektor fuer das Jacobi-Verfahren. */
void get_u0 (double *u0, int N, char fill_type)
{
        // Zufaelliger Startvektor
        if (fill_type == 'r')
        {
                get_random_vec(u0,N*N,0,1);
        }
        // Testvektor nur aus Einsen
        else if (fill_type == 't')
        {
                for (int i = 0; i < N*N; i++)
                {
                        u0[i] = 1;
                }
        }
}


/////////////////////////////////////////////////////////////////
//                      Kernel-Funktionen                      //
/////////////////////////////////////////////////////////////////


/* Kernel-Funktion zur eintragsweisen einmaligen Jacobi-Iteration.
 Jeder Thread berechnet einen Eintrag der neuen Iterierten. */
__global__ void jacobi (double *u_ghost, double *f, int N)
{

    // Statische Allokation des Shared Memory; Daten von einem Teilgitter sollen geladen werden
    __shared__ double u_ghost_loc[blocksize*blocksize];
    __shared__ double f_loc[(blocksize-2)*(blocksize-2)];


    // Bestimmung der Indizes des Vektoreintrags, der zu aufrufendem Thread korrespondiert
    int k_ghost = blockIdx.y*(blocksize-2)*(N+2) + threadIdx.y*(N+2) + blockIdx.x*(blocksize-2) + threadIdx.x ; // Globaler Index auf Gitter mit Ghost Layer
    int k_ghost_loc = threadIdx.y*blocksize + threadIdx.x; // Lokaler Index auf Gitter mit Ghost Layer

    int k = (blockIdx.y*(blocksize-2) + threadIdx.y-1)*N + blockIdx.x*(blocksize-2) + threadIdx.x-1; // Globaler Index auf inneren Gitterpunkten
    int k_loc = (threadIdx.y-1)*(blocksize-2) + threadIdx.x-1; // Lokaler Index auf inneren Gitterpunkten

    // Achtung: Die Betrachtung von k und k_loc ist nur fuer Threads sinnvoll, die tatsaechlich zu inneren Gitterpunkten korrespondieren!


    // Lade Daten von Teilgitter in Shared Memory; jeder Thread uebernimmt einen Eintrag
    u_ghost_loc[k_ghost_loc] = u_ghost[k_ghost]; // Eintraege der vorherigen Iterierten
    if ( (k_ghost_loc >= blocksize) && (k_ghost_loc%blocksize != 0) && (k_ghost_loc%blocksize != blocksize-1) && (k_ghost_loc < (blocksize-1)*blocksize) )
    {
        f_loc[k_loc] = f[k]; // Eintraege der rechten Seite werden nur fuer innere Gitterpunkte geladen
    }

    __syncthreads();


    // Kernel: Eintragsweise Jacobi-Iteration; nur Threads zu inneren Gitterpunkten des Teilgitters rechnen; Ergebnis wird direkt in Global Memory geschrieben
    if ( (k_ghost_loc >= blocksize) && (k_ghost_loc%blocksize != 0) && (k_ghost_loc%blocksize != blocksize-1) && (k_ghost_loc < (blocksize-1)*blocksize) )
    {
        u_ghost[k_ghost] = (f_loc[k_loc] + u_ghost_loc[k_ghost_loc-blocksize] + u_ghost_loc[k_ghost_loc-1] + u_ghost_loc[k_ghost_loc+1] + u_ghost_loc[k_ghost_loc+blocksize])/4;
    }

}


/* Kernel-Funktion zur eintragsweisen Bestimmung des Residuums einer Iterierten des Jacobi-Verfahrens.
 Jeder Thread berechnet einen Eintrag des Residuums. */
__global__ void residual (double *res, double *u_ghost, double *f, int N)
{

    // Statische Allokation des Shared Memory; Daten von einem Teilgitter sollen geladen werden
    __shared__ double u_ghost_loc[blocksize*blocksize];
    __shared__ double f_loc[(blocksize-2)*(blocksize-2)];


    // Bestimmung der Indizes des Vektoreintrags, der zu aufrufendem Thread korrespondiert
    int k_ghost = blockIdx.y*(blocksize-2)*(N+2) + threadIdx.y*(N+2) + blockIdx.x*(blocksize-2) + threadIdx.x ; // Globaler Index auf Gitter mit Ghost Layer
    int k_ghost_loc = threadIdx.y*blocksize + threadIdx.x; // Lokaler Index auf Gitter mit Ghost Layer

    int k = (blockIdx.y*(blocksize-2) + threadIdx.y-1)*N + blockIdx.x*(blocksize-2) + threadIdx.x-1; // Globaler Index auf inneren Gitterpunkten
    int k_loc = (threadIdx.y-1)*(blocksize-2) + threadIdx.x-1; // Lokaler Index auf inneren Gitterpunkten

    // Achtung: Die Betrachtung von k und k_loc ist nur fuer Threads sinnvoll, die tatsaechlich zu inneren Gitterpunkten korrespondieren!


    // Lade Daten von Teilgitter in Shared Memory; jeder Thread uebernimmt einen Eintrag
    u_ghost_loc[k_ghost_loc] = u_ghost[k_ghost]; // Eintraege der vorherigen Iterierten
    if ( (k_ghost_loc >= blocksize) && (k_ghost_loc%blocksize != 0) && (k_ghost_loc%blocksize != blocksize-1) && (k_ghost_loc < (blocksize-1)*blocksize) )
    {
        f_loc[k_loc] = f[k]; // Eintraege der rechten Seite werden nur fuer innere Gitterpunkte geladen
    }

    __syncthreads();


    // Kernel: Eintragsweise Berechnung des Residuums; nur Threads zu inneren Gitterpunkten des Teilgitters rechnen; Ergebnis wird direkt in Global Memory geschrieben
    if ( (k_ghost_loc >= blocksize) && (k_ghost_loc%blocksize != 0) && (k_ghost_loc%blocksize != blocksize-1) && (k_ghost_loc < (blocksize-1)*blocksize) )
    {
        res[k] = (- u_ghost_loc[k_ghost_loc-blocksize] - u_ghost_loc[k_ghost_loc-1] + 4*u_ghost_loc[k_ghost_loc] - u_ghost_loc[k_ghost_loc+1] -u_ghost_loc[k_ghost_loc+blocksize]) - f_loc[k_loc];
    }

}



/* Kernel-Funktion zur punktweisen multiplikation zweier Vektoren */
__global__ void pmult (double *v1, double *v2)
{
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    //Ergebnis liegt nun in v1
    v1[i]*=v2[i];
}



/* Kernel-Funktion zur Reduktion bzw. Fan-in */
__global__ void reduction (double *iv, double *ov)
{
    // Statische Allokation des Shared Memory; Groesse des Blocks
    __shared__ double sv[blocksize-2];

    //Hole Daten in shared memory
    int tid = threadIdx.x;
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    sv[tid]=iv[i];
    __syncthreads();

    //Reduktion in shared memory auf Block durchfuehren
    for ( int s=1; s<blockDim.x; s*=2)
    {
                // Nur gerade Threads rechnen
                if (tid%(2*s)==0)
                {
                        //Wenn Zugriff innerhalb des Vektors erfolgt
                        if (tid+s<blocksize-2)
                        {
                                sv[tid]+=sv[tid+s];
                        }
                }
                __syncthreads();
        }

        //Ergebnis herausschreiben
        if(tid==0)
        {
                ov[blockIdx.x]=sv[0];
        }

}
/* Optimierte Kernel-Funktion zur Reduktion bzw. Fan-in */
__global__ void reduction2 (double *iv, double *ov)
{
    // Statische Allokation des Shared Memory; Groesse des Blocks
    __shared__ double sv[blocksize];

    //Hole Daten in shared memory
    int tid = threadIdx.x;

    // Direkte Addition zweier Bloecke
    int i = blockIdx.x*(blockDim.x*2)+threadIdx.x;
    sv[tid]=iv[i]+iv[i+blockDim.x];
    __syncthreads();

    //Reduktion in shared memory auf Block durchfuehren
    for ( int s=blockDim.x/2; s>0; s>>=1)
    {
                if (tid<s)
                {
                        //Wenn Zugriff innerhalb des Vektors erfolgt
                        if (tid+s<blocksize)
                        {
                                sv[tid]+=sv[tid+s];
                        }
                }
                __syncthreads();
        }

        //Ergebnis herausschreiben
        if(tid==0)
        {
                ov[blockIdx.x]=sv[0];
        }
}

/* Kernel-Funktion zur Bestimmung der Norm eines Vektors auf den inneren Gitterpunkten.
__global__ void norm (double *v, int N)
{
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    //Ergebnis liegt nun in v
    v[i]*=v[i];
    __syncthreads();
    int temp = v[0];
    v[0]=0;
    v[0]+=v[i];
    v[0]+=temp;

}*/



////////////////////////////////////////////////////////////////
//                        Main-Funktion                        //
/////////////////////////////////////////////////////////////////


int main()
{

    printf("\n*******************************************\n");
    printf("*      Jacobi-Verfahren auf der GPU       *\n");
    printf("*******************************************\n\n");



    // Deklaration benoetigter Variablen

    int N; // Problemgroesse
    int max_iter; // Maximale Iterationszahl
    double rel_tol; // Relative Toleranz
    char fill_type; // Art der Befuellung des Startvektors

    double *u; // Iterierte des Jacobi-Verfahrens
    double *u_ghost, *dev_u_ghost; // Iterierte des Jacobi-Verfahrens auf Gitter mit Ghost Layer
    double *f, *dev_f; // Rechte Seite
    double *sol; // Exakte Loesung

    double *dev_res; // Residuum => Nur auf GPU benoetigt!
    double *dev_res_init; // Initiales Residuum => Nur auf GPU benoetigt!

    double *norm_res, *dev_norm_res; // Residuumsnorm
    double *norm_res_init, *dev_norm_res_init; // Residuumsnorm des initalen Residuums

    int iter; // Iterationszahl

    // Zeitnahme
    clock_t start;
    clock_t stop;
    clock_t difference;
    int msec;



    // Nutzereingaben

    printf("Problemgroesse (Vielfaches von %d):  ",blocksize-2);
    scanf("%d",&N);

    // Achtung: N%(blocksize-2) == 0 erforderlich!

    printf("Maximale Iterationszahl: ");
    scanf("%d",&max_iter);

    printf("Relative Toleranz: ");
    scanf("%lf",&rel_tol);

    printf("Startvektor (r,t): ");
    scanf("%c",&fill_type);
    fill_type = getchar();

    printf("\n");



    // Beginn der Zeitnahme
    start = clock();



    // Speicherallokation und Befuellung der Vektoren auf der CPU

    printf("Erstelle Daten auf CPU ...\n");

    // Iterierte
    u = (double *)malloc(N*N*sizeof(double));
    get_u0(u,N,fill_type);

    // Iterierte auf Gitter mit Ghost Layer
    u_ghost = (double *)malloc((N+2)*(N+2)*sizeof(double));
    ghostify(u,u_ghost,N);

    // Rechte Seite
    f = (double *)malloc(N*N*sizeof(double));
    get_RHS(f,N);

    // Exakte Loesung
    sol = (double *)malloc(N*N*sizeof(double));
    get_solution(sol,N);

    // Residuumsnorm
    norm_res = (double *)malloc(sizeof(double));

    // Residuumsnorm des initialen Residuums
    norm_res_init = (double *)malloc(sizeof(double));




    // Speicherallokation im globalen Speicher der GPU

    printf("Allokiere Speicher auf GPU ...\n");

    // Iterierte auf Gitter mit Ghost Layer
    hipMalloc((void**)&dev_u_ghost,(N+2)*(N+2)*sizeof(double));

    // Rechte Seite
    hipMalloc((void**)&dev_f,N*N*sizeof(double));

    // Residuum
    hipMalloc((void**)&dev_res,N*N*sizeof(double));

    // Initiales Residuum
    hipMalloc((void**)&dev_res_init,N*N*sizeof(double));

    // Residuumsnorm
    hipMalloc((void**)&dev_norm_res,sizeof(double));

    // Residuumsnorm des initialen Residuums
    hipMalloc((void**)&dev_norm_res_init,sizeof(double));



    // Kopieren benoetigter Daten in den globalen Speicher der GPU

    printf("Kopiere Daten auf GPU ...\n");

    // Iterierte auf Gitter mit Ghost Layer
    hipMemcpy(dev_u_ghost,u_ghost,(N+2)*(N+2)*sizeof(double),hipMemcpyHostToDevice);

    // Rechte Seite
    hipMemcpy(dev_f,f,N*N*sizeof(double),hipMemcpyHostToDevice);



    // Baue Gitter von Blocks fuer die GPU-Threads; 2D-Gitter zur Erfassung eines Teilgitters der Diskretisierung

    printf("Baue Gitter fuer die GPU ...\n");

    int nblocks = N/(blocksize-2); // Anzahl Blocks pro Gitterdimension
    dim3 gridDim (nblocks,nblocks);
    dim3 blockDim (blocksize,blocksize);

    // Ggf. neues Gitter zur Normbestimmung ...
    //Bestimme Anzahl rekursiver Aufrufe fuer den Redukton-Kernel
        int l=1;
        int NN=N;
        while(NN>blocksize-2)
        {
                    l+=1;
                    NN=NN/(blocksize-2);
                    // Fuer Reduction2:
                    //NN=NN/(2*blocksize);
            }

            //Anzahl Bloecke je Level
            int *nb=(int*)malloc(l*sizeof(double));
            nb[0]=N/(blocksize-2);
            for (int i=0; i<l; i++)
            {
                    nb[i]=nb[i-1]/(blocksize-2);
                    // Fuer Reduction2:
                    //nb[i]=nb[i-1]/(2*blocksize);
            }

            //Zwei Arrays fuer Ergebnisse
            double *dev_r1;
            hipMalloc((void**)&dev_r1,nb[0]*sizeof(double));
            double *dev_r2;
            hipMalloc((void**)&dev_r2,nb[0]*sizeof(double));


        // Ausfuehrung des Jacobi-Verfahrens auf der GPU

        printf("Jacobi-Iterationen ...\n");

        // Bestimmung der Norm des initialen Residuums fuer die relative Abbruchbedingung
        residual<<<gridDim,blockDim>>>(dev_res_init,dev_u_ghost,dev_f,N);
        // norm <<<...,...>>>(dev_res_init,N);

        // Berechnen der Norm

        pmult<<<nb[0],blocksize-2>>>(dev_res_init,dev_res_init);
        // Fuer Reduction2:
        // pmult<<<nb[0]*2,blocksize>>>(dev_res_init,dev_res_init);

        //Reduktion 0, dev_res_init wir blockweise in dev_r1 reduziert
        reduction<<<nb[0],blocksize-2>>>(dev_res_init,dev_r1);
        // Fuer Reduction2:
        // reduction2<<<nb[0],blocksize>>>(dev_res_init,dev_r1);

        //Restliche Reduktionen, immer weniger Bloecke
        //Abwechselnd ist r1 Input und r2 Output und umgekehrt
        for (int i=1; i<l; i++)
        {
                    if (i%2==1)
                    {
                            reduction<<<nb[i],blocksize-2>>>(dev_r1,dev_r2);
                    }
                    else
                    {
                            reduction<<<nb[i],blocksize-2>>>(dev_r2,dev_r1);
                    }
            }

            // Fuer Reduction2:
            //Restliche Reduktionen, immer weniger Bloecke
        //Abwechselnd ist r1 Input und r2 Output und umgekehrt
        /*for (int i=1; i<l; i++)
        {
                    if (nb[i]>0)
                    {
                            if (i%2==1)
                            {
                                    reduction2<<<nb[0],blocksize>>>(dev_r1,dev_r2);
                            }
                            else
                            {
                                    reduction2<<<nb[i],blocksize>>>(dev_r2,dev_r1);
                            }
                    }
                    else
                    {
                            if (i%2==1)
                            {
                                    reduction2<<<nb[i-1]/2,blocksize>>>(dev_r1,dev_r2);
                            }
                            else
                            {
                                    reduction2<<<nb[i-1]/2,blocksize>>>(dev_r2,dev_r1);
                            }
                    }
            }*/


        // Ergebnis von GPU zu Host kopieren
        if (l%2==1)
            {
                    hipMemcpy(norm_res_init,dev_r1,sizeof(double),hipMemcpyDeviceToHost);
            }
            else
            {
                    hipMemcpy(norm_res_init,dev_r2,sizeof(double),hipMemcpyDeviceToHost);
            }
            printf("Residuumsnorm: %e\n", *norm_res_init);
            //Wurzelziehen vom Eintrag des Vektors, der auf die CPU kopiert wurde
            //sq(norm_res_init);

        // hipMemcpy(norm_res_init,dev_norm_res_init,sizeof(double),hipMemcpyDeviceToHost);

        // Jacobi-Schleife
        iter = 0;
        while (true)
        {
                // Ausfuehrung der Jacobi-Iteration
                jacobi<<<gridDim,blockDim>>>(dev_u_ghost,dev_f,N);
                iter += 1;

                // Bestimmung der Residuumsnorm
                residual<<<gridDim,blockDim>>>(dev_res,dev_u_ghost,dev_f,N);
                // norm <<<...,...>>>(dev_res,N);

                pmult<<<nb[0],blocksize-2>>>(dev_res,dev_res);
                // Fuer Reduction2:
                        // pmult<<<nb[0]*2,blocksize>>>(dev_res_init,dev_res_init);


                        //Reduktion 0, dev_res_init wir blockweise in dev_r1 reduziert
                        reduction<<<nb[0],blocksize-2>>>(dev_res,dev_r1);
                        // Fuer Reduction2:
                        // reduction2<<<nb[0],blocksize>>>(dev_res,dev_r1);


                        //Restliche Reduktionen, immer weniger Bloecke
                        //Abwechselnd ist r1 Input und r2 Output und umgekehrt
                        for (int i=1; i<l; i++)
                        {
                                if (i%2==1)
                                {
                                        reduction<<<nb[i],blocksize-2>>>(dev_r1,dev_r2);
                                }
                                else
                                {
                                        reduction<<<nb[i],blocksize-2>>>(dev_r2,dev_r1);
                                }
                        }

                        // Fuer Reduction2:
                        //Restliche Reduktionen, immer weniger Bloecke
                        //Abwechselnd ist r1 Input und r2 Output und umgekehrt
                        /*for (int i=1; i<l; i++)
                        {
                                if (nb[i]>0)
                                {
                                        if (i%2==1)
                                        {
                                                reduction2<<<nb[i],blocksize>>>(dev_r1,dev_r2);
                                        }
                                        else
                                        {
                                                reduction2<<<nb[i],blocksize>>>(dev_r2,dev_r1);
                                        }
                                }
                                else
                                {
                                        if (i%2==1)
                                        {
                                                reduction2<<<nb[i-1]/2,blocksize>>>(dev_r1,dev_r2);
                                        }
                                        else
                                        {
                                                reduction2<<<nb[i-1]/2,blocksize>>>(dev_r2,dev_r1);
                                        }
                                }
                        }*/


                        // Ergebnis von GPU zu Host kopieren
                        if (l%2==1)
                        {
                                hipMemcpy(norm_res,dev_r1,sizeof(double),hipMemcpyDeviceToHost);
                        }
                        else
                        {
                                hipMemcpy(norm_res,dev_r2,sizeof(double),hipMemcpyDeviceToHost);
                        }
                        //Wurzelziehen vom Eintrag des Vektors, der auf die CPU kopiert wurde
                        //sq(norm_res);
                // hipMemcpy(norm_res_init,dev_norm_res_init,sizeof(double),hipMemcpyDeviceToHost);

                // Abbruchbedingung
                if (iter >= max_iter || sqrt((*norm_res))/sqrt((*norm_res_init)) < rel_tol)
                {
                    break;
                }
            }



            // Kopieren des Ergebnisses von globalem Speicher der GPU zurueck auf die CPU

            printf("Kopiere Ergebnis auf CPU ...\n");

            // Iterierte auf Gitter mit Ghost Layer
            hipMemcpy(u_ghost,dev_u_ghost,(N+2)*(N+2)*sizeof(double),hipMemcpyDeviceToHost);
            deghostify(u_ghost,u,N);

            printf("\n  => Berechnungen abgeschlossen!\n\n");



            // Ende der Zeitnahme
            stop = clock();
            difference = stop-start;
            msec = difference*1000/CLOCKS_PER_SEC;



            // Ausgabe der Resultate

            printf("Absoluter Fehler: %e\n", get_abs_err(u,sol,N*N));
            printf("Residuumsnorm: %e\n", *norm_res);
            printf("Iterationen: %d\n", iter);
            printf("Benoetigte Zeit: %d.%ds\n\n", msec/1000,msec%1000);
            // Speicherfreigabe auf der CPU
                free(u);
                free(u_ghost);
                free(f);
                free(sol);
                free(norm_res);
                free(norm_res_init);
                free(nb);



                // Speicherfreigabe auf der GPU
                hipFree(dev_u_ghost);
                hipFree(dev_f);
                hipFree(dev_res);
                hipFree(dev_res_init);
                hipFree(norm_res);
                hipFree(norm_res_init);
                hipFree(dev_r1);
                hipFree(dev_r2);



                return 0;
            }
