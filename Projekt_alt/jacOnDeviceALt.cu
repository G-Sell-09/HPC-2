
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>



__global__ void residual(float* x_now, float* x_next, float* b, int N)
{
    int j;
    float sigma = 0.0;
    int row = blockIdx.x*blockDim.x+threadIdx.x;


    if (row < N)
    {
      printf(" %d ", row);
      if (row == 0)
      {
        sigma += 4*x_now[0] - x_now[N] - x_now[1]; //bottom left

      }
      else if (row == N-1)
      {
        sigma += 4*x_now[(N-1)*N] - x_now[(N-1)*N + 1] - x_now[(N-2)*N]; //top left

        for (j=1; j<N-1;j++)
          sigma += 4*x_now[(N-1)*N + j] - x_now[(N-1)*N + j-1] - x_now[(N-1)*N + j+1] - x_now[(N-2)*N + j]; //top edge

        sigma += 4*x_now[(N-1)*N + N-1] - x_now[(N-1)*N + N-2] - x_now[(N-2)*N + N-1]; //top right
      }
      else
      {
        sigma += 4*x_now[row*N] - x_now[row*N + 1] - x_now[(row-1)*N] - x_now[(row+1)*N]; //left edge (i = row)

        for (j=1; j<N-1;j++)
          sigma += 4*x_now[row*N + j] - x_now[row*N + j-1] - x_now[row*N + j+1] - x_now[(row-1)*N + j] - x_now[(row+1)*N + j]; // inner points (i=row)

        sigma += 4*x_now[row*N + N-1]   - x_now[row*N + N-2]     - x_now[(row-1)*N + N-1] - x_now[(row+1)*N + N-1]; // right edge
      }


    x_next[row] = sigma;
  }
}


// Kernel Funktion fuer die Durchfuehrung einer Jacobi-Iteration
__global__ void jacobiOnDevice(float* x_now, float* x_next, float* b, int N)
{
    int j;
    float sigma = 0.0;
    float omega = 0.6;
    float aDiag = 0.25;
    int row = blockIdx.y*blockDim.y+threadIdx.y;


    if (row < N)
    {
      if (row == 0)
      {
        sigma += 4*x_now[0] - x_now[N] - x_now[1]; //bottom left

        for (j=1; j<N-1;j++)
          sigma += 4*x_now[j] - x_now[j-1] - x_now[j+1] - x_now[N + j];  // bottom edge

        sigma += 4*x_now[N-1] - x_now[2*N-1] - x_now[N - 2];	//bottom right
      }
      else if (row == N-1)
      {
        sigma += 4*x_now[(N-1)*N] - x_now[(N-1)*N + 1] - x_now[(N-2)*N]; //top left

        for (j=1; j<N-1;j++)
          sigma += 4*x_now[(N-1)*N + j] - x_now[(N-1)*N + j-1] - x_now[(N-1)*N + j+1] - x_now[(N-2)*N + j]; //top edge

        sigma += 4*x_now[(N-1)*N + N-1] - x_now[(N-1)*N + N-2] - x_now[(N-2)*N + N-1]; //top right
      }
      else
      {
        sigma += 4*x_now[row*N] - x_now[row*N + 1] - x_now[(row-1)*N] - x_now[(row+1)*N]; //left edge (i = row)

        for (j=1; j<N-1;j++)
          sigma += 4*x_now[row*N + j] - x_now[row*N + j-1] - x_now[row*N + j+1] - x_now[(row-1)*N + j] - x_now[(row+1)*N + j]; // inner points (i=row)

        sigma += 4*x_now[row*N + N-1]   - x_now[row*N + N-2]     - x_now[(row-1)*N + N-1] - x_now[(row+1)*N + N-1]; // right edge
      }


    x_next[row] = aDiag * omega *(b[row] - sigma);
  }
}



int main(int argc, char* argv[]){


    int N = atoi(argv[1]);
    int iter = 10;
    int k;
    clock_t before = clock();

    float *x_now = (float*)malloc(N*sizeof(float));
    float *x_next = (float*)malloc(N*sizeof(float));
    float *b = (float*)malloc(N*sizeof(float));
    float res, dev_res;

    float *dev_x_now,*dev_x_next,*dev_b;


    //Allokiere Speicher im globalen Speicher der GPU
    hipMalloc((void**)&dev_x_now,N*sizeof(float));
    hipMalloc((void**)&dev_x_next,N*sizeof(float));
    hipMalloc((void**)&dev_b,N*sizeof(float));

    //Füllen der Arrays auf der CPU
    for (int i=0;i<N;i++)
    {
      x_now[i] = 1.0;
      x_next[i] = 0.0;
      b[i] = 1.0;
    }

    //Kopiere Daten auf GPU in globalen Speicher
    hipMemcpy(dev_x_now,x_now,N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(dev_x_next,x_next,N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(dev_b,b,N*sizeof(float),hipMemcpyHostToDevice);

    //Baue 2D Gitter von Blocks der Größe 32x32 Threads
    int nblocks = (N+32)/32;
    //dim3 gridDim(nblocks,nblocks);
    //dim3 blockDim(32,32);

    //Aufruf des Jacobi Verfahrens auf der GPU
    // for (k=0; k<iter; k++)
    // {
    //     if (k%2)
    //         jacobiOnDevice<<<gridDim,blockDim>>>(dev_x_next, dev_x_now, dev_b, N);
    //     else
    //         jacobiOnDevice<<<gridDim,blockDim>>>(dev_x_now, dev_x_next, dev_b, N);
    // }

    residual<<<nblocks,32>>>(dev_x_now, dev_x_next, dev_b, N);

    //Ergebnis zurück auf den Host kopieren
    // if (k%2)
    //   cudaMemcpy(x_next,dev_x_now,N*sizeof(float),cudaMemcpyDeviceToHost);
    // else
    //   cudaMemcpy(x_next,dev_x_next,N*sizeof(float),cudaMemcpyDeviceToHost);

    hipMemcpy(x_next,dev_x_next, N*sizeof(float),hipMemcpyDeviceToHost);


    clock_t after = clock();
    clock_t difference = clock() - before;
    int msec = difference * 1000 / CLOCKS_PER_SEC;

    printf("\nTime taken %d.%d seconds \n\n",msec/1000,msec%1000);

    printf("Result: C= ");

    for (int i=0;i<N;i++){
        printf(" %f",x_next[i]);
    }

    hipFree(dev_x_now);
    hipFree(dev_x_next);
    hipFree(dev_b);

    free(x_now);
    free(x_next);
    free(b);

    return 0;
}
